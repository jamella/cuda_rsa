#include "hip/hip_runtime.h"
#include "gmp_location.h"
#include "main.h"
#include "io.h"
#include "gcd.h"
#include "gcdCuda.h"

void dispatchGcdCalls(u1024bit_t *array, uint32_t *found, int count, FILE *dfp, FILE *nfp) {

   // resultant bit vector on host
   uint8_t bitVector[NUM_BLOCKS];

   // pointers on device
   u1024bit_t *d_keys;
   u1024bit_t *d_currentKey;
   uint8_t *d_bitVector;

   dim3 blockDim(BLOCK_DIM_X, BLOCK_DIM_Y);
   dim3 gridDim(GRID_DIM_X, GRID_DIM_Y);

   // allocate space for current key, keys to compare and bit vector
   /*OLD*/
   /*HANDLE_ERROR(hipMalloc((void **) &d_currentKey,
      sizeof(u1024bit_t)));
   HANDLE_ERROR(hipMalloc((void **) &d_keys,
      sizeof(u1024bit_t) * BLOCK_DIM_Y * NUM_BLOCKS));
   HANDLE_ERROR(hipMalloc((void **) &d_bitVector,
      sizeof(uint8_t) * NUM_BLOCKS));*/

    /*NEW*/
   HANDLE_ERROR(hipMalloc((void **) &d_currentKey,
      sizeof(u1024bit_t) * count));
   HANDLE_ERROR(hipMalloc((void **) &d_keys,
      sizeof(u1024bit_t) * count));
   HANDLE_ERROR(hipMalloc((void **) &d_bitVector,
      sizeof(uint8_t) * NUM_BLOCKS));

         // copy current key
         HANDLE_ERROR(hipMemcpy(d_currentKey, array,
            sizeof(u1024bit_t) * count,
            hipMemcpyHostToDevice));

d_keys = d_currentKey;
         // copy list of keys
         /*HANDLE_ERROR(hipMemcpy(d_keys, array,
            sizeof(u1024bit_t) * count,
            hipMemcpyHostToDevice));*/




   int i;
   int j;
   int stride = NUM_BLOCKS * BLOCK_DIM_Y;

    /*Transfer the entire key space here*/


    /*Then invoke the loop. Treat i,j as a pointer offsets.*/


    /*There shouldn't need to be much in the way of changes to the kernel.
    Because the kernel already works with pointer offsets anyway, and since
    we always know where we are in the key space from the host side, we can
    simply pass the kernel a pointer offset, and that way we keep the kernel
    from having to do any funky indexing or anything else.*/

   for (i = 0; i < count; i++) {
      for (j = i + 1; j < count; j += stride) {
         // copy current key
   /*      HANDLE_ERROR(hipMemcpy(d_currentKey, array + i,
            sizeof(u1024bit_t),
            hipMemcpyHostToDevice));*/

         // copy list of keys
         /*toCopy = j + stride >= count ? count - j : stride;

         HANDLE_ERROR(hipMemcpy(d_keys, array + j,
            sizeof(u1024bit_t) * toCopy,
            hipMemcpyHostToDevice));*/

         // initialize bit vector to 0
         HANDLE_ERROR(hipMemset(d_bitVector, 0,
            sizeof(uint8_t) * NUM_BLOCKS));

         // kernel call
         cuGCD<<<gridDim, blockDim>>>(d_currentKey + i, d_keys + j, d_bitVector);

         HANDLE_ERROR(hipPeekAtLastError());

         // copy bit vector back
         HANDLE_ERROR(hipMemcpy(bitVector, d_bitVector,
            sizeof(uint8_t) * NUM_BLOCKS,
            hipMemcpyDeviceToHost));

         computeAndOutputGCDs(array, found, bitVector, i, j, dfp, nfp);
      }
   }

   // do freeing
   hipFree(d_keys);
   hipFree(d_currentKey);
   hipFree(d_bitVector);
}

__global__ void cuGCD(u1024bit_t *key, u1024bit_t *key_comparison_list, 
   uint8_t *bitvector) {

   __shared__ u1024bit_t shkey[BLOCK_DIM_Y * GRID_DIM_X];
   __shared__ u1024bit_t compareto_keys[BLOCK_DIM_Y];

    /* We are using blocks of size (x, y) (32, 8),
    so each row in a block will be responsible for computing one set of
    key comparisons */

   int keyNum = (BLOCK_DIM_Y * blockIdx.x) + threadIdx.y;
   int index = threadIdx.x;

   int i;
   for (i = 0; i < BLOCK_DIM_Y * GRID_DIM_X; i++) {
      shkey[i].number[index] = key->number[index];
   }
   for(i = 0; i < BLOCK_DIM_X; i++){
        compareto_keys[threadIdx.y].number[i] = key_comparison_list[keyNum].number[i];

   }

   __syncthreads();

    /*OLD*/
   /*gcd(shkey[keyNum].number, key_comparison_list[keyNum].number);*/

   gcd(shkey[keyNum].number, compareto_keys[threadIdx.y].number);

    /*OLD*/
   /*if (isGreaterThanOne(key_comparison_list[keyNum].number)) {
      bitvector[keyNum / 8] |= LOW_ONE_MASK << (keyNum % 8);
   }*/

   if (isGreaterThanOne(compareto_keys[threadIdx.y].number)) {
      bitvector[keyNum / 8] |= LOW_ONE_MASK << (keyNum % 8);
   }
}

// result ends up in y; x is also overwritten
__device__ void gcd(unsigned int *x, unsigned int *y) {
   int c = 0;

   if (isNonZero(x) && isNonZero(y)) {

      while (((x[WORDS_PER_KEY - 1] | y[WORDS_PER_KEY - 1]) & 1) == 0) {
         shiftR1(x);
         shiftR1(y);
         c++;
      }

      while (__any(x[threadIdx.x])) {

         while ((x[WORDS_PER_KEY - 1] & 1) == 0) {
            shiftR1(x);
         }

         while ((y[WORDS_PER_KEY - 1] & 1) == 0) {
            shiftR1(y);
         }

         if (geq(x, y)) {
            subtract(x, y);
            shiftR1(x);
         }
         else {
            subtract(y, x);
            shiftR1(y);
         }
      }

      shiftL(y, c);
   }
   else if (isNonZero(y)) {
      y[threadIdx.x] = x[threadIdx.x];
   }
}

__device__ void shiftR1(unsigned int *arr)
{
   unsigned int index = threadIdx.x;
   uint32_t temp = 0;

   if (index != 0)
   {
      temp = arr[index - 1];
   }

   arr[index] >>= 1;
   arr[index] |= (temp << WORDS_PER_KEY - 1);
}

__device__ void shiftL1(unsigned int *arr)
{
   unsigned int index = threadIdx.x;
   uint32_t temp = 0;

   if (index != WORDS_PER_KEY - 1)
   {
      temp = arr[index + 1];
   }

   arr[index] <<= 1;
   arr[index] |= (temp >> WORDS_PER_KEY - 1);
}

__device__ void shiftL(unsigned int *arr, unsigned int x) {
   int i;
   for (i = 0; i < x; i++) {
      shiftL1(arr);
   }
}

__device__ void subtract(uint32_t *x, uint32_t *y) {
   __shared__ uint8_t borrow[BLOCK_DIM_Y][WORDS_PER_KEY];
   uint8_t *borrowPtr = borrow[threadIdx.y];

   uint8_t index = threadIdx.x;

   if (index == 0) {
      borrowPtr[WORDS_PER_KEY - 1] = 0;
   }

   unsigned int temp;
   temp = x[index] - y[index];

   if (index > 0) {
      borrowPtr[index - 1] = (temp > x[index]);
   }

   while (__any(borrowPtr[index])) {
      if (borrowPtr[index]) {
         temp--;
      }

      if (index > 0) {
         borrowPtr[index - 1] = (temp == 0xffffffffU && borrow[index]);
      }
   }

   x[index] = temp;
}

__device__ int geq(uint32_t *x, uint32_t *y) {
   __shared__ unsigned int pos[BLOCK_DIM_Y];

   int index = threadIdx.x;

   if (index == 0) {
      pos[threadIdx.y] = WORDS_PER_KEY - 1;
   }

   if (x[index] != y[index]) {
      atomicMin(&pos[threadIdx.y], index);
   }

   return x[pos[threadIdx.y]] >= y[pos[threadIdx.y]];
}

__device__ int isNonZero(uint32_t *x) {
   __shared__ uint8_t nonZeroFound[BLOCK_DIM_Y];

   uint8_t index = threadIdx.x;

   if (index == 0) {
      nonZeroFound[threadIdx.y] = 0;
   }

   if (x[index] != 0) {
      nonZeroFound[threadIdx.y] = 1;
   }

   return nonZeroFound[threadIdx.y];
}

__device__ int isGreaterThanOne(uint32_t *number) {
   __shared__ uint8_t greaterThanOne[BLOCK_DIM_Y];

   uint8_t index = threadIdx.x;

   if (index == 0) {
      greaterThanOne[threadIdx.y] = 0;
   }

   if (index < WORDS_PER_KEY - 1 && number[index] > 0) {
      greaterThanOne[threadIdx.y] = 1;
   }
   else if (index == WORDS_PER_KEY - 1 && number[index] > 1) {
      greaterThanOne[threadIdx.y] = 1;
   }

   return greaterThanOne[threadIdx.y];
}

static void HandleError( hipError_t err,
    const char *file,
    int line ) {
  if (err != hipSuccess) {
    printf( "%s in %s at line %d\n", hipGetErrorString( err ),
        file, line );
    exit( EXIT_FAILURE );
  }
}
